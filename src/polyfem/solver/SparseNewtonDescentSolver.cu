#include "hip/hip_runtime.h"
#pragma once

#include "SparseNewtonDescentSolver.hpp"

#include "polyfem/utils/CUDA_utilities.cuh"
#include "polyfem/utils/CuSparseUtils.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "hipsparse.h"
#include "hip/library_types.h"

namespace cppoptlib
{
	template <typename ProblemType>
	bool SparseNewtonDescentSolver<ProblemType>::check_direction_gpu(
		const polyfem::StiffnessMatrix &hessian,
		const Eigen::Matrix<double, -1, 1> &grad,
		const Eigen::Matrix<double, -1, 1> &direction)
	{
		// gradient descent, check descent direction
		// const double residual = (hessian * direction + grad).norm(); // H Δx + g = 0
		// const double residual_cpu = (hessian * direction + grad).norm(); // H Δx + g = 0

		POLYFEM_SCOPED_TIMER("checking direction", this->checking_direction_time);
		int N = hessian.cols();

		double *hessian_dev, *direction_dev, *grad_dev, *tmp_dev, *res_dev; // to compute residual
		double *grad_grad_dev, *grad_direc_dev;                             // to compute grad norm and grad dot direction

		// move hessian to gpu (compressed format)
		const int non0 = hessian.nonZeros();
		polyfem::logger().trace("non0: {}, cols: {}, rows: {}, allocating size: {} bytes", non0, hessian.cols(), hessian.rows(), non0 * sizeof(double));
		int *row_dev, *col_dev;
		// row_dev = ALLOCATE_GPU<int>(row_dev, (N+1)*sizeof(int));
		// col_dev = ALLOCATE_GPU<int>(col_dev, non0*sizeof(int));
		// hessian_dev = ALLOCATE_GPU<double>(hessian_dev, non0*sizeof(double));
		EigenSparseToCuSparseTranspose(hessian, row_dev, col_dev, hessian_dev);

		// compute residual
		// const double residual = (hessian * direction + grad).norm(); // H Δx + g = 0
		tmp_dev = ALLOCATE_GPU<double>(tmp_dev, N * sizeof(double));
		COPYDATATOGPU<double>(tmp_dev, grad.data(), N * sizeof(double));
		residual_dev = ALLOCATE_GPU<double>(residual_dev, sizeof(double));

		// double *hessian_host = hessian.valuePtr();
		const double *direction_host = direction.data();
		const double *grad_host = grad.data();
		double *tmp_host = new double[N];
		double *res_host = new double[1];
		double *grad_grad_host = new double[1];
		double *grad_direc_host = new double[1];

		double alpha = 1.0;
		double beta = 1.0; // 0.0 for cublas

		// printf("before allocating hessian_dev:\n");
		// check_cuda_mem();

		const int non0 = hessian.nonZeros();
		polyfem::logger().trace("non0: {}, cols: {}, rows: {}, allocating size: {} bytes", non0, hessian.cols(), hessian.rows(), non0 * sizeof(double));
		// std::cout << "non0: " << non0 << " , cols: "<< hessian.cols() <<  ", rows: " << hessian.rows() << ", allocating size : " << non0*sizeof(double) << " bytes" << std::endl;
		int *row_dev;
		int *col_dev;
		row_dev = ALLOCATE_GPU<int>(row_dev, (N + 1) * sizeof(int));
		col_dev = ALLOCATE_GPU<int>(col_dev, non0 * sizeof(int));
		hessian_dev = ALLOCATE_GPU<double>(hessian_dev, non0 * sizeof(double));
		EigenSparseToCuSparseTranspose(hessian, row_dev, col_dev, hessian_dev);

		int *row_host = new int[N + 1];
		int *col_host = new int[non0];
		double *hessian_host = new double[non0];
		COPYDATATOHOST<int>(row_host, row_dev, (N + 1) * sizeof(int));
		COPYDATATOHOST<int>(col_host, col_dev, (non0) * sizeof(int));
		COPYDATATOHOST<double>(hessian_host, hessian_dev, (non0) * sizeof(double));
		int base = row_host[N] - non0;

		// printf("hessian:\n");
		// std::cout << MatrixXd(hessian) << std::endl;
		// polyfem::logger().trace("row_host[N](none zero elements): {}", row_host[N]);
		// printf("\nrow_host[N]: %d\n", row_host[N]);
		// printf("cusparse index base: %d\n", base);
		// printf("\ncol_host:\n");
		// for(int i=0;i<non0;i++)
		//     printf("%d, ",col_host[i]);
		// printf("\nrow_host:\n");
		// for(int i=0;i<N+1;i++)
		//     printf("%d, ",row_host[i]);
		// printf("\nhessian_host:\n");
		// for(int i=0;i<non0;i++)
		//     printf("%lf, ",hessian_host[i]);

		// printf("\nafter allocating and copying hessian_dev:\n");
		// check_cuda_mem();

		// hessian_dev = ALLOCATE_GPU<double>(hessian_dev, N*N*sizeof(double));
		direction_dev = ALLOCATE_GPU<double>(direction_dev, N * sizeof(double));
		grad_dev = ALLOCATE_GPU<double>(grad_dev, N * sizeof(double));
		tmp_dev = ALLOCATE_GPU<double>(tmp_dev, N * sizeof(double));
		res_dev = ALLOCATE_GPU<double>(res_dev, sizeof(double));
		grad_grad_dev = ALLOCATE_GPU<double>(grad_grad_dev, sizeof(double));
		grad_direc_dev = ALLOCATE_GPU<double>(grad_grad_dev, sizeof(double));

		// COPYDATATOGPU<double>(hessian_dev, hessian_host, N*N*sizeof(double));
		COPYDATATOGPU<double>(direction_dev, direction_host, N * sizeof(double));
		COPYDATATOGPU<double>(grad_dev, grad_host, N * sizeof(double));

		// hipblasHandle_t handle;
		// hipblasCreate(&handle);

		// hipDeviceSynchronize();
		// hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, N, N, &alpha, direction_dev, 1, hessian_dev, N, &beta, tmp_dev, 1);  // hessian * direction
		// hipDeviceSynchronize();
		// hipblasDaxpy(handle, N, &alpha, grad_dev, 1, tmp_dev, 1); // vector add, hessian * direction + grad
		// hipDeviceSynchronize();
		// hipblasDdot(handle, N, tmp_dev, 1, tmp_dev, 1, res_dev);  // dot product
		// hipDeviceSynchronize();

		////
		COPYDATATOGPU<double>(tmp_dev, grad_host, N * sizeof(double));

		hipsparseStatus_t status;
		hipsparseHandle_t handle = 0;
		hipsparseMatDescr_t descr = 0;
		status = hipsparseCreate(&handle);
		status = hipsparseCreateMatDescr(&descr);
		hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
		hipsparseIndexBase_t cubase;
		hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
		double *buffer;
		buffer = ALLOCATE_GPU<double>(buffer, 2 * non0 * sizeof(double));
		status = cusparseCsrmvEx(handle, CUSPARSE_ALG_MERGE_PATH, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, non0, &alpha, HIP_R_64F, descr, hessian_dev, HIP_R_64F, row_dev, col_dev, direction_dev, HIP_R_64F, &beta, HIP_R_64F, tmp_dev, HIP_R_64F, HIP_R_64F, buffer);
		hipsparseDestroyMatDescr(descr);
		hipsparseDestroy(handle);
		hipFree(row_dev);
		hipFree(col_dev);
		hipFree(buffer);

		hipblasHandle_t handle2;
		hipblasCreate(&handle2);
		hipblasDdot(handle2, N, tmp_dev, 1, tmp_dev, 1, res_dev); // dot product
		COPYDATATOHOST<double>(res_host, res_dev, sizeof(double));
		const double residual = std::sqrt(*res_host); // norm

		// polyfem::logger().trace("residual_cpu: {}, residual_gpu: {}, diff: {}", residual_cpu, residual, residual-residual_cpu);
		////

		// hipblasHandle_t handle2;
		// hipblasCreate(&handle2);

		hipblasDdot(handle2, N, grad_dev, 1, grad_dev, 1, grad_grad_dev);
		hipblasDdot(handle2, N, grad_dev, 1, direction_dev, 1, grad_direc_dev);
		COPYDATATOHOST<double>(grad_grad_host, grad_grad_dev, sizeof(double));
		COPYDATATOHOST<double>(grad_direc_host, grad_direc_dev, sizeof(double));
		const double grad_norm = std::sqrt(*grad_grad_host);
		const double grad_direc_prod = *grad_direc_host;

		delete[] res_host;
		delete[] tmp_host;
		delete[] grad_grad_host;
		delete[] grad_direc_host;
		hipFree(hessian_dev);
		hipFree(direction_dev);
		hipFree(grad_dev);
		hipFree(tmp_dev);
		hipFree(res_dev);
		hipblasDestroy(handle2);

		// gradient descent, check descent direction

		if (std::isnan(residual))
		{
			increase_descent_strategy();
			polyfem::logger().log(
				this->descent_strategy == 2 ? spdlog::level::warn : spdlog::level::debug,
				"nan linear solve residual {} (||∇f||={}); reverting to {}",
				residual, grad_norm, this->descent_strategy_name());
			return false;
		}
		else if (residual > std::max(1e-8 * grad_norm, 1e-5))
		{
			increase_descent_strategy();
			polyfem::logger().log(
				this->descent_strategy == 2 ? spdlog::level::warn : spdlog::level::debug,
				"large linear solve residual {} (||∇f||={}); reverting to {}",
				residual, grad_norm, this->descent_strategy_name());
			return false;
		}
		else
		{
			polyfem::logger().trace("linear solve residual {}", residual);
		}

		// do this check here because we need to repeat the solve without resetting reg_weight
		if (grad_direc_prod >= 0)
		{
			increase_descent_strategy();
			polyfem::logger().log(
				this->descent_strategy == 2 ? spdlog::level::warn : spdlog::level::debug,
				"[{}] direction is not a descent direction (Δx⋅g={}≥0); reverting to {}",
				name(), direction.dot(grad), descent_strategy_name());
			return false;
		}

		return true;
	}
	template class SparseNewtonDescentSolver<polyfem::solver::NLProblem>;
	template class SparseNewtonDescentSolver<polyfem::solver::FullNLProblem>;
} // namespace cppoptlib